#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <string>

#include "graph.h"
#include "bfsCPU.h"
#include "bfsCUDA.cu"
#include "hwtimer.h"

#define GPU_DEVICE 0

void runCpu(int startVertex, Graph &G, std::vector<int> &distance,
            std::vector<int> &parent, std::vector<bool> &visited) {
    bfsCPU(startVertex, G, distance, parent, visited);
}

void checkError(hipError_t  error, std::string msg) {
    if (error != hipSuccess) {
        printf("%s: %d\n", msg.c_str(), error);
        exit(1);
    }
}

hipDeviceProp_t deviceProp;

int* d_adjacencyList;
int* d_edgesOffset;
int* d_edgesSize;
int* d_distance;
int* d_parent;
int* d_currentQueue;
int* d_nextQueue;
int* d_degrees;
int* incrDegrees;

void initCuda(Graph &G) {
    //initialize CUDA
    checkError(hipGetDeviceProperties(&deviceProp, GPU_DEVICE), "cannot get device");
    printf("setting device %d with name %s\n", GPU_DEVICE, deviceProp.name);
    checkError(hipSetDevice(GPU_DEVICE), "cannot set device");

    //copy memory to device
    checkError(hipMalloc(&d_adjacencyList, G.numEdges * sizeof(int)), "cannot allocate d_adjacencyList");
    checkError(hipMalloc(&d_edgesOffset, G.numVertices * sizeof(int)), "cannot allocate d_edgesOffset");
    checkError(hipMalloc(&d_edgesSize, G.numVertices * sizeof(int)), "cannot allocate d_edgesSize");
    checkError(hipMalloc(&d_distance, G.numVertices * sizeof(int)), "cannot allocate d_distance");
    checkError(hipMalloc(&d_parent, G.numVertices * sizeof(int)), "cannot allocate d_parent");
    checkError(hipMalloc(&d_currentQueue, G.numVertices * sizeof(int)), "cannot allocate d_currentQueue");
    checkError(hipMalloc(&d_nextQueue, G.numVertices * sizeof(int)), "cannot allocate d_nextQueue");
    checkError(hipMalloc(&d_degrees, G.numVertices * sizeof(int)), "cannot allocate d_degrees");
    checkError(hipHostMalloc((void **) &incrDegrees, sizeof(int) * G.numVertices), "cannot allocate memory");

    checkError(hipMemcpy(d_adjacencyList, G.adjacencyList.data(), G.numEdges * sizeof(int), hipMemcpyHostToDevice),
               "cannot copy to d_adjacencyList");
    checkError(hipMemcpy(d_edgesOffset, G.edgesOffset.data(), G.numVertices * sizeof(int), hipMemcpyHostToDevice),
               "cannot copy to d_edgesOffset");
    checkError(hipMemcpy(d_edgesSize, G.edgesSize.data(), G.numVertices * sizeof(int), hipMemcpyHostToDevice),
               "cannot copy to d_edgesSize");
}

void finalizeCuda() {
    //free memory
    checkError(hipFree(d_adjacencyList), "cannot free memory for d_adjacencyList");
    checkError(hipFree(d_edgesOffset), "cannot free memory for d_edgesOffset");
    checkError(hipFree(d_edgesSize), "cannot free memory for d_edgesSize");
    checkError(hipFree(d_distance), "cannot free memory for d_distance");
    checkError(hipFree(d_parent), "cannot free memory for d_parent");
    checkError(hipFree(d_currentQueue), "cannot free memory for d_parent");
    checkError(hipFree(d_nextQueue), "cannot free memory for d_parent");
    checkError(hipHostFree(incrDegrees), "cannot free memory for incrDegrees");
}

void checkOutput(std::vector<int> &distance, std::vector<int> &expectedDistance, Graph &G) {
    for (int i = 0; i < G.numVertices; i++) {
        if (distance[i] != expectedDistance[i]) {
            printf("%d %d %d\n", i, distance[i], expectedDistance[i]);
            printf("Wrong output!\n");
            exit(1);
        }
    }

    printf("Output OK!\n\n");
}

void initializeCudaBfs(int startVertex, std::vector<int> &distance, std::vector<int> &parent, Graph &G) {
    //initialize values
    std::fill(distance.begin(), distance.end(), std::numeric_limits<int>::max());
    std::fill(parent.begin(), parent.end(), std::numeric_limits<int>::max());
    distance[startVertex] = 0;
    parent[startVertex] = 0;

    checkError(hipMemcpy(d_distance, distance.data(), G.numVertices * sizeof(int), hipMemcpyHostToDevice),
               "cannot copy to d)distance");
    checkError(hipMemcpy(d_parent, parent.data(), G.numVertices * sizeof(int), hipMemcpyHostToDevice),
               "cannot copy to d_parent");

    int firstElementQueue = startVertex;
    hipMemcpy(d_currentQueue, &firstElementQueue, sizeof(int), hipMemcpyHostToDevice);
}

void finalizeCudaBfs(std::vector<int> &distance, std::vector<int> &parent, Graph &G) {
    //copy memory from device
    checkError(hipMemcpy(distance.data(), d_distance, G.numVertices * sizeof(int), hipMemcpyDeviceToHost),
               "cannot copy d_distance to host");
    checkError(hipMemcpy(parent.data(), d_parent, G.numVertices * sizeof(int), hipMemcpyDeviceToHost), "cannot copy d_parent to host");

}

void runCudaBfs(int startVertex, Graph &G, std::vector<int> &distance,
                      std::vector<int> &parent) {
    initializeCudaBfs(startVertex, distance, parent, G);

    int *changed;
    checkError(hipHostMalloc((void **) &changed, sizeof(int)), "cannot allocate changed");

    //launch kernel
    *changed = 1;
    int level = 0;

    while (*changed) {
        *changed = 0;
        dim3 grid(G.numVertices / 512 + 1, 1, 1);
        dim3 block(512, 1, 1);

        simpleBfs<<<grid, block>>>(G.numVertices, level, d_adjacencyList, d_edgesOffset, d_edgesSize, d_distance, d_parent, changed);
       
        hipDeviceSynchronize();
        level++;
    }

    finalizeCudaBfs(distance, parent, G);
}

int main(int argc, char **argv) {
    if (argc != 4) {
        printf("usage: ./bfs-exec <start vertex> <number of vertices> <number of edges>\n");
        exit(1);
    }

    hwtimer_t timer;
    initTimer(&timer);
    
    // read graph from standard input
    Graph G;
    int startVertex = atoi(argv[1]);

    readGraph(G, argc, argv);

    printf("Number of vertices %d\n", G.numVertices);
    printf("Number of edges %d\n\n", G.numEdges);

    //vectors for results
    std::vector<int> distance(G.numVertices, std::numeric_limits<int>::max());
    std::vector<int> parent(G.numVertices, std::numeric_limits<int>::max());
    std::vector<bool> visited(G.numVertices, false);
 
    //run CPU sequential bfs
    printf("Starting sequential bfs.\n");
    startTimer(&timer);
    runCpu(startVertex, G, distance, parent, visited);
    stopTimer(&timer);
    printf("Elapsed time: %lld ns.\n\n", getTimerNs(&timer));

    //save results from sequential bfs
    std::vector<int> expectedDistance(distance);
    std::vector<int> expectedParent(parent);

    //run CUDA simple parallel bfs
    initCuda(G);
    printf("Starting parallel bfs.\n");
    startTimer(&timer);
    runCudaBfs(startVertex, G, distance, parent);
    stopTimer(&timer);
    printf("Elapsed time: %lld ns.\n\n", getTimerNs(&timer));

    checkOutput(distance, expectedDistance, G);

    finalizeCuda();
    return 0;
}
